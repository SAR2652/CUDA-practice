#include "hip/hip_runtime.h"
#include <stdio.h>
#include <limits.h>
#include <random>
#include <cstdlib>
#include <algorithm>
#include "cuda_utils.h"

__global__ void reduceMinMax(int* input, int* output_min, int* output_max,
                             int N)
{
    // You can allocate only one shared variable
    extern __shared__ int sdata[];

    // Therefore allocate 2X dynamic shared memory size and manually split it
    int* sdata_min = sdata;
    int* sdata_max = sdata + blockDim.x;

    int tid = threadIdx.x;
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Assign all values in a block to shared memory
    sdata_min[tid] = (i < N) ? input[i]: INT_MAX;
    sdata_max[tid] = (i < N) ? input[i]: INT_MIN;

    __syncthreads();

    for(unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if(tid < stride)
        {
            sdata_min[tid] = min(sdata_min[tid], sdata_min[tid + stride]);
            sdata_max[tid] = max(sdata_max[tid], sdata_max[tid + stride]);
        }
        __syncthreads();
    }

    if(tid == 0)
    {
        output_min[blockIdx.x] = sdata_min[0];
        output_max[blockIdx.x] = sdata_max[0];
    }
}


int main()
{
    int N = 1 << 20;
    size_t size = N * sizeof(int);

    int *h_in = new int[N];

    std::random_device rd;
    std::mt19937 gen(rd());
    
    for(int i = 0 ; i < N; i++)
    {
        h_in[i] = gen();
    }

    int threadsPerBlock = 256;
    int dynamicSharedMemorySize = threadsPerBlock * sizeof(int);

    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    int *h_out = new int[blocksPerGrid];

    int *d_in, *d_out_min, *d_out_max;
    CHECK_CUDA(hipMalloc((void **) &d_in, size));
    CHECK_CUDA(hipMalloc((void **) &d_out_min, blocksPerGrid * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **) &d_out_max, blocksPerGrid * sizeof(int)));

    CHECK_CUDA(hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice));
    reduceMinMax<<<blocksPerGrid, threadsPerBlock,
    2 * dynamicSharedMemorySize>>>(d_in, d_out_min, d_out_max, N);

    int *h_out_min = new int[blocksPerGrid];
    int *h_out_max = new int[blocksPerGrid];

    CHECK_CUDA(hipMemcpy(h_out_min, d_out_min, blocksPerGrid * sizeof(int),
               hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_out_max, d_out_max, blocksPerGrid * sizeof(int),
               hipMemcpyDeviceToHost));

    int minimum = INT32_MAX;
    int maximum = INT32_MIN;

    for(int i = 0; i < blocksPerGrid; i++)
    {
        if(h_out_min[i] < minimum)
        {
            minimum = h_out_min[i];
        }

        if(h_out_max[i] > maximum)
        {
            maximum = h_out_max[i];
        }
    }

    printf("Minimum = %d\n", minimum);
    printf("Maximum = %d\n", maximum);

    return 0;
}